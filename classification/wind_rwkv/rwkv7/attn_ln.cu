#include "hip/hip_runtime.h"
// Copyright (c) 2024, Johan Sokrates Wind

#include "tile.cuh"
#include <assert.h>
typedef bf * __restrict__ F_;
typedef float * __restrict__ F32_;

constexpr int WARPS = _C_/16;
constexpr int fw_stages = 1, bw_stages = 1;

__device__ FTile fw_post_process(const FTile&x0, float*params, const FTile&q, const FTile&k, const FTile&v, const FTile&g, float*share) {
    constexpr int C = _C_;
    float2 mean = sumh<WARPS>(x0, (float*)share);
    constexpr float iC = 1.f/C;
    mean.x *= iC;
    mean.y *= iC;
    FTile center;
    for (int i = 0; i < 8; i++) center.fdata[i] = x0.fdata[i]-(i/2%2?mean.y:mean.x);
    FTile sq;
    for (int i = 0; i < 8; i++) sq.fdata[i] = center.fdata[i]*center.fdata[i];
    float2 var = sumh<WARPS>(sq, (float*)share);
    float2 istd = {rsqrtf(var.x*iC+64e-5f), rsqrtf(var.y*iC+64e-5f)};

    int j0 = threadIdx.x%4*2+threadIdx.x/32*16, j1 = j0+8;
    float ln_w[4] = {params[j0], params[j0+1], params[j1], params[j1+1]}, ln_b[4] = {params[C+j0], params[C+j0+1], params[C+j1], params[C+j1+1]};
    FTile x1;
    for (int i = 0; i < 8; i++) x1.fdata[i] = center.fdata[i]*(i/2%2?istd.y:istd.x) * ln_w[i%2+i/4*2] + ln_b[i%2+i/4*2];

    float cur_bonus[4] = {params[C*2+j0], params[C*2+j0+1], params[C*2+j1], params[C*2+j1+1]};
    FTile qk_bonus;
    for (int i = 0; i < 8; i++)
        qk_bonus.fdata[i] = q.fdata[i]*k.fdata[i]*cur_bonus[i%2+i/4*2];
    float2 dot = sumh<WARPS>(qk_bonus, (float*)share);
    for (int i = 0; i < 8; i++) x1.fdata[i] += (i/2%2?dot.y:dot.x) * v.fdata[i];

    return x1*g;
}

//TODO: why is this (somewhat) slow? Probably because of excessive __syncthreads()
__device__ RTile bw_post_process(const FTile&x0, float*params, const FTile&q, const FTile&k, const FTile&v, const FTile&g, const RTile&dout, 
        float*dparams, FTile&dq, FTile&dk, FTile&dv, FTile&dg, float*share) {
    constexpr int C = _C_;
    float2 mean = sumh<WARPS>(x0, (float*)share);
    constexpr float iC = 1.f/C;
    mean.x *= iC;
    mean.y *= iC;
    FTile center;
    for (int i = 0; i < 8; i++) center.fdata[i] = x0.fdata[i]-(i/2%2?mean.y:mean.x);
    FTile sqr;
    for (int i = 0; i < 8; i++) sqr.fdata[i] = center.fdata[i]*center.fdata[i];
    float2 var = sumh<WARPS>(sqr, (float*)share);
    float2 istd = {rsqrtf(var.x*iC+64e-5f), rsqrtf(var.y*iC+64e-5f)};

    int j0 = threadIdx.x%4*2+threadIdx.x/32*16, j1 = j0+8;
    float ln_w[4] = {params[j0], params[j0+1], params[j1], params[j1+1]}, ln_b[4] = {params[C+j0], params[C+j0+1], params[C+j1], params[C+j1+1]};
    FTile x1;
    for (int i = 0; i < 8; i++) x1.fdata[i] = center.fdata[i]*(i/2%2?istd.y:istd.x) * ln_w[i%2+i/4*2] + ln_b[i%2+i/4*2];

    float cur_bonus[4] = {params[C*2+j0], params[C*2+j0+1], params[C*2+j1], params[C*2+j1+1]};
    FTile qk_bonus;
    for (int i = 0; i < 8; i++)
        qk_bonus.fdata[i] = q.fdata[i]*k.fdata[i]*cur_bonus[i%2+i/4*2];
    float2 dot = sumh<WARPS>(qk_bonus, (float*)share);
    for (int i = 0; i < 8; i++) x1.fdata[i] += (i/2%2?dot.y:dot.x) * v.fdata[i];

    dg = (FTile)dout * x1;
    FTile dx1 = dout * g;

    for (int i = 0; i < 8; i++) dv.fdata[i] = (i/2%2?dot.y:dot.x) * dx1.fdata[i];

    float2 ddot = sumh<WARPS>(dx1*v, (float*)share);

    for (int i = 0; i < 8; i++) {
        float cb = cur_bonus[i%2+i/4*2], ddoti = (i/2%2?ddot.y:ddot.x);
        dq.fdata[i] = k.fdata[i]*cb*ddoti;
        dk.fdata[i] = q.fdata[i]*cb*ddoti;
        dparams[8+i%2+i/4*2] += k.fdata[i]*q.fdata[i]*ddoti;
    }

    dparams[4] += dx1.fdata[0]+dx1.fdata[2];
    dparams[5] += dx1.fdata[1]+dx1.fdata[3];
    dparams[6] += dx1.fdata[4]+dx1.fdata[6];
    dparams[7] += dx1.fdata[5]+dx1.fdata[7];

    FTile dcenter, dvar;
    for (int i = 0; i < 8; i++) {
        float istdi = (i/2%2?istd.y:istd.x), ln_wi = ln_w[i%2+i/4*2];
        dcenter.fdata[i] = dx1.fdata[i] * istdi * ln_wi;
        dparams[i%2+i/4*2] += dx1.fdata[i] * istdi * center.fdata[i];
        dvar.fdata[i] = dcenter.fdata[i] * center.fdata[i];
    }

    float2 dvar2 = sumh<WARPS>(dvar, (float*)share);
    for (int i = 0; i < 8; i++) {
        float istdi = (i/2%2?istd.y:istd.x), dvar2i = (i/2%2?dvar2.y:dvar2.x);
        dcenter.fdata[i] -= dvar2i*iC * center.fdata[i] * istdi * istdi;
    }
    float2 dcenter_mean = sumh<WARPS>(dcenter, (float*)share);
    for (int i = 0; i < 8; i++) {
        float dcenter_meani = (i/2%2?dcenter_mean.y:dcenter_mean.x);
        dcenter.fdata[i] -= dcenter_meani*iC;
    }
    return (RTile)dcenter;
}

__global__ void forward_kernel(int T, int H, F_ w_, F_ q_, F_ k_, F_ v_, F_ a_, F_ b_, F_ g_, F32_ params_, F_ s0_, bf* y_, bf* s_, bf* sT_) {
    constexpr int C = _C_, K = 16;
    int bi = blockIdx.y, hi = blockIdx.x;
    extern __shared__ char smem_[];
    char*smem = smem_;

    STile *sw_ = (STile*)smem; smem += sizeof(STile)*fw_stages*WARPS;
    STile *sq_ = (STile*)smem; smem += sizeof(STile)*fw_stages*WARPS;
    STile *sk_ = (STile*)smem; smem += sizeof(STile)*fw_stages*WARPS;
    STile *sv_ = (STile*)smem; smem += sizeof(STile)*fw_stages*WARPS;
    STile *sa_ = (STile*)smem; smem += sizeof(STile)*fw_stages*WARPS;
    STile *sb_ = (STile*)smem; smem += sizeof(STile)*fw_stages*WARPS;
    STile *sg_ = (STile*)smem; smem += sizeof(STile)*fw_stages*WARPS;
    char*share = (char*)smem;

    int stride = H*C;
    int warpi = threadIdx.x/32;

    auto push = [&](int t) {
        int off = bi*T*H*C + t*K*H*C + hi*C + warpi*16;
        int si = t%fw_stages;
        sw_[si*WARPS+warpi] = GTile(w_+off, stride);
        sq_[si*WARPS+warpi] = GTile(q_+off, stride);
        sk_[si*WARPS+warpi] = GTile(k_+off, stride);
        sv_[si*WARPS+warpi] = GTile(v_+off, stride);
        sa_[si*WARPS+warpi] = GTile(a_+off, stride);
        sb_[si*WARPS+warpi] = GTile(b_+off, stride);
        sg_[si*WARPS+warpi] = GTile(g_+off, stride);
    };
    for (int t = 0; t < fw_stages-1 && t < T/K; t++) push(t), __commit_group();

    FTile state[WARPS];
    for (int i = 0; i < WARPS; i++) {
        int off = bi*H*C*C + hi*C*C + warpi*16*C + i*16;
        RTile tmp;
        tmp = GTile(s0_+off, C);
        state[i] = tmp;
    }

    __shared__ float params[C*3];
    if (threadIdx.x < C) {
        params[threadIdx.x] = params_[hi*C+threadIdx.x];
        params[C+threadIdx.x] = params_[H*C+hi*C+threadIdx.x];
        params[C*2+threadIdx.x] = params_[H*C*2+hi*C+threadIdx.x];
    }

    for (int t = 0; t < T/K; t++) {
        __syncthreads();
        if (t+fw_stages-1 < T/K)
            push(t+fw_stages-1);
        __commit_group();
        __wait_groups<fw_stages-1>();
        __syncthreads();
        int si = t%fw_stages;
        STile &sw = sw_[si*WARPS+warpi], &sq = sq_[si*WARPS+warpi], &sk = sk_[si*WARPS+warpi], &sv = sv_[si*WARPS+warpi], &sa = sa_[si*WARPS+warpi], &sb = sb_[si*WARPS+warpi], &sg = sg_[si*WARPS+warpi];

        FTile w = (RTile)sw;
        apply_(w, [](float x) { return __expf(-__expf(x)); });
        FTile fw = w;
        FTile non_incl_pref = cumprodv<0,0>(fw);
        FTile incl_pref = non_incl_pref * w;
        FTile inv_incl_pref = incl_pref;
        apply_(inv_incl_pref, [](float x) { return 1.f/x; });

        RTile wq = (RTile)sq *     incl_pref, kwi = (RTile)sk * inv_incl_pref;
        RTile wa = (RTile)sa * non_incl_pref, bwi = (RTile)sb * inv_incl_pref;
        FTile ab = sum_warp<1,WARPS>((float*)share, tril<1>(wa % bwi));
        RTile ak = sum_warp<1,WARPS>((float*)share, tril<1>(wa % kwi));

        RTile ab_inv;
        __syncthreads();
        if (threadIdx.x < 32) ab_inv = tri_minv(ab, (float*)share);
        __syncthreads();
        ab_inv = from_warp(ab_inv, 0, (float4*)share);

        RTile vt = sv.t();
        FTile ab_ut = vt % ak;
        for (int i = 0; i < WARPS; i++)
            ab_ut += state[i] % from_warp(wa, i, (float4*)share);
        RTile ut = FTile(ab_ut % ab_inv);

        FTile y = sum_warp<1,WARPS>((float*)share, tril<0>(wq % kwi)) % vt;
        y +=      sum_warp<1,WARPS>((float*)share, tril<0>(wq % bwi)) % ut;
        for (int i = 0; i < WARPS; i++)
            y += from_warp(wq, i, (float4*)share) % state[i];

        y = fw_post_process(y, params, (RTile)sq, (RTile)sk, (RTile)sv, (RTile)sg, (float*)share);

        int off = bi*T*H*C + t*K*H*C + hi*C + warpi*16;
        GTile(y_+off, stride) = RTile(y);

        RTile kwt = transpose(kwi*fw), bwt = transpose(bwi*fw);
        for (int i = 0; i < WARPS; i++) {
            int off = bi*H*(T/K)*C*C + hi*(T/K)*C*C + t*C*C + warpi*16*C + i*16;
            GTile(s_+off, C) = (RTile)state[i];

            FTile fstate = state[i] * from_warp(fw, i, (float4*)share);
            fstate += vt % from_warp(kwt, i, (float4*)share);
            fstate += ut % from_warp(bwt, i, (float4*)share);
            state[i] = fstate;
        }
    }
    for (int i = 0; i < WARPS; i++) {
        int off = bi*H*C*C + hi*C*C + warpi*16*C + i*16;
        GTile(sT_+off, C) = state[i];
    }
}

void cuda_forward(int B, int T, int H, bf*w, bf*q, bf*k, bf*v, bf*a, bf*b, bf*g, float*params, bf*s0, bf*y, bf*s, bf*sT) {
    assert(T%16 == 0);
    constexpr int tmp_size1 = sizeof(float)*32*8*WARPS, tmp_size2 = sizeof(float)*16*16*2;
    constexpr int threads = 32*WARPS, shared_mem = sizeof(STile)*fw_stages*WARPS*7 + (tmp_size1 > tmp_size2 ? tmp_size1 : tmp_size2);
    static int reported = 0;
    if (!reported++) {
#if defined VERBOSE
        printf("forward_kernel() uses %d bytes of (dynamic) shared memory\n", shared_mem);
#endif
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(forward_kernel));
        int cur_mem = attr.maxDynamicSharedSizeBytes;
        if (shared_mem > cur_mem) {
#if defined VERBOSE
            printf("Increasing forward_kernel's MaxDynamicSharedMemorySize from %d to %d\n", cur_mem, shared_mem);
#endif
            assert(!hipFuncSetAttribute(reinterpret_cast<const void*>(forward_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem));
        }
    }
    forward_kernel<<<dim3(H,B), dim3(threads), shared_mem>>>(T,H,w,q,k,v,a,b,g,params,s0,y,s,sT);
}


__global__ void backward_kernel(int T, int H, F_ w_, F_ q_, F_ k_, F_ v_, F_ a_, F_ b_, F_ g_, F32_ params_, F_ dy_, F_ s_, F_ dsT_, bf* dw_, bf* dq_, bf* dk_, bf* dv_, bf* da_, bf* db_, bf* dg_, float* dparams_, bf* ds0_) {
    constexpr int C = _C_, K = 16;
    int bi = blockIdx.y, hi = blockIdx.x;
    extern __shared__ char smem_[];
    char*smem = smem_;

    STile *sw_ = (STile*)smem; smem += sizeof(STile)*bw_stages*WARPS;
    STile *sq_ = (STile*)smem; smem += sizeof(STile)*bw_stages*WARPS;
    STile *sk_ = (STile*)smem; smem += sizeof(STile)*bw_stages*WARPS;
    STile *sv_ = (STile*)smem; smem += sizeof(STile)*bw_stages*WARPS;
    STile *sa_ = (STile*)smem; smem += sizeof(STile)*bw_stages*WARPS;
    STile *sb_ = (STile*)smem; smem += sizeof(STile)*bw_stages*WARPS;
    STile *sg_ = (STile*)smem; smem += sizeof(STile)*bw_stages*WARPS;
    STile *sdy_ = (STile*)smem; smem += sizeof(STile)*bw_stages*WARPS;
    STile *state_ = (STile*)smem; smem += sizeof(STile)*bw_stages*WARPS*WARPS;
    char*share = (char*)smem;

    int stride = H*C;
    int warpi = threadIdx.x/32;

    auto push = [&](int t) {
        int off = bi*T*H*C + t*K*H*C + hi*C + warpi*16;
        int si = t%bw_stages;
        sw_[si*WARPS+warpi] = GTile(w_+off, stride);
        sq_[si*WARPS+warpi] = GTile(q_+off, stride);
        sk_[si*WARPS+warpi] = GTile(k_+off, stride);
        sv_[si*WARPS+warpi] = GTile(v_+off, stride);
        sa_[si*WARPS+warpi] = GTile(a_+off, stride);
        sb_[si*WARPS+warpi] = GTile(b_+off, stride);
        sg_[si*WARPS+warpi] = GTile(g_+off, stride);
        sdy_[si*WARPS+warpi] = GTile(dy_+off, stride);
        for (int i = 0; i < WARPS; i++) {
            int off2 = bi*H*(T/K)*C*C + hi*(T/K)*C*C + t*C*C + warpi*16*C + i*16;
            state_[si*WARPS*WARPS+warpi*WARPS+i] = GTile(s_+off2, C);
        }
    };

    FTile dstate[WARPS];
    for (int i = 0; i < WARPS; i++) {
        int off = bi*H*C*C + hi*C*C + warpi*16*C + i*16;
        RTile tmp;
        tmp = GTile(dsT_+off, C);
        dstate[i] = tmp;
        __commit_group();
    }

    __shared__ float params[C*3];
    float dparams[12] = {};
    if (threadIdx.x < C) {
        params[threadIdx.x] = params_[hi*C+threadIdx.x];
        params[C+threadIdx.x] = params_[H*C+hi*C+threadIdx.x];
        params[C*2+threadIdx.x] = params_[H*C*2+hi*C+threadIdx.x];
    }

    for (int t = 0; t < bw_stages-1 && t < T/K; t++) push(T/K-1-t), __commit_group();

    for (int t = T/K-1; t >= 0; t--) {
        __syncthreads();
        if (t-bw_stages+1 >= 0)
            push(t-bw_stages+1);
        __commit_group();
        __wait_groups<bw_stages-1>();
        __syncthreads();
        int si = t%bw_stages;
        STile &sw = sw_[si*WARPS+warpi], &sq = sq_[si*WARPS+warpi], &sk = sk_[si*WARPS+warpi], &sv = sv_[si*WARPS+warpi], &sa = sa_[si*WARPS+warpi], &sb = sb_[si*WARPS+warpi], &sg = sg_[si*WARPS+warpi], &sdy = sdy_[si*WARPS+warpi];
        STile*state = state_+si*WARPS*WARPS;

        FTile w = (RTile)sw;
        apply_(w, [](float x) { return __expf(-__expf(x)); });
        FTile fw = w;
        FTile non_incl_pref = cumprodv<0,0>(fw);
        FTile incl_pref = non_incl_pref * w;
        FTile inv_incl_pref = incl_pref;
        apply_(inv_incl_pref, [](float x) { return 1.f/x; });

        RTile wq = (RTile)sq *     incl_pref, kwi = (RTile)sk * inv_incl_pref;
        RTile wa = (RTile)sa * non_incl_pref, bwi = (RTile)sb * inv_incl_pref;
        FTile ab = sum_warp<1,WARPS>((float*)share, tril<1>(wa % bwi));
        RTile ak = sum_warp<1,WARPS>((float*)share, tril<1>(wa % kwi));

        RTile ab_inv;
        __syncthreads();
        if (threadIdx.x < 32) ab_inv = tri_minv(ab, (float*)share);
        __syncthreads();
        ab_inv = from_warp(ab_inv, 0, (float4*)share);

        RTile vt = sv.t();
        FTile ab_ut = vt % ak;
        for (int i = 0; i < WARPS; i++)
            ab_ut += state[warpi*WARPS+i] % from_warp(wa, i, (float4*)share);
        RTile ut = FTile(ab_ut % ab_inv);

        FTile y = sum_warp<1,WARPS>((float*)share, tril<0>(wq % kwi)) % vt;
        y +=      sum_warp<1,WARPS>((float*)share, tril<0>(wq % bwi)) % ut;
        for (int i = 0; i < WARPS; i++)
            y += from_warp(wq, i, (float4*)share) % state[warpi*WARPS+i];

        FTile dq, dk, dv, dg;
        //dq.zero_();
        //dk.zero_();
        //dv.zero_();
        //dg.zero_();
        RTile dyt = transpose(bw_post_process(y, params, (RTile)sq, (RTile)sk, (RTile)sv, (RTile)sg, (RTile)sdy, dparams, dq, dk, dv, dg, (float*)share));
        int off = bi*T*H*C + t*K*H*C + hi*C + warpi*16;
        GTile(dg_+off, stride) = RTile(dg);

        RTile qb = sum_warp<1,WARPS>((float*)share, tril<0>(wq % bwi));
        RTile qk = sum_warp<1,WARPS>((float*)share, tril<0>(wq % kwi));

        FTile dut = FTile(dyt % transpose(qb));
        dv += transpose(qk) % dyt;
        for (int i = 0; i < WARPS; i++) {
            RTile dstatei = dstate[i];
            dut += dstatei % from_warp(bwi*fw, i, (float4*)share);
            dv += from_warp(kwi*fw, i, (float4*)share) % dstatei;
        }
        RTile dab_ut = FTile(dut % transpose(ab_inv));
        dv += transpose(ak) % dab_ut;

        GTile(dv_+off, stride) = RTile(dv);

        FTile dab = sum_warp<1,WARPS>((float*)share, tril<1>(transpose(dab_ut) % transpose(ut)));
        FTile dak = sum_warp<1,WARPS>((float*)share, tril<1>(transpose(dab_ut) % transpose(vt)));
        FTile dab_u_state0;
        dab_u_state0.zero_();
        for (int i = 0; i < WARPS; i++)
            dab_u_state0 += from_warp(transpose(dab_ut), i, (float4*)share) % state[i*WARPS+warpi].t();

        FTile da = dab_u_state0;
        da += dab % transpose(bwi);
        da += dak % transpose(kwi);
        da = non_incl_pref * da;
        GTile(da_+off, stride) = RTile(da);

        FTile dqb = sum_warp<1,WARPS>((float*)share, tril<0>(transpose(dyt) % transpose(ut)));
        FTile dqk = sum_warp<1,WARPS>((float*)share, tril<0>(transpose(dyt) % transpose(vt)));
        FTile dy_state0;
        dy_state0.zero_();
        for (int i = 0; i < WARPS; i++)
            dy_state0 += from_warp(transpose(dyt), i, (float4*)share) % state[i*WARPS+warpi].t();

        FTile dq2 = dy_state0;
        dq2 += dqb % transpose(bwi);
        dq2 += dqk % transpose(kwi);
        dq += incl_pref * dq2;
        GTile(dq_+off, stride) = RTile(dq);

        RTile wqt = transpose(wq), wat = transpose(wa);

        FTile u_dstate, v_dstate, dw;
        u_dstate.zero_();
        v_dstate.zero_();
        dw.zero_();
        RTile ones;
        for (int i = 0; i < 4; i++) ones.data[i] = to_bf2({1.f,1.f});
        for (int i = 0; i < WARPS; i++) {
            int tid = threadIdx.x%32;
            if (warpi == i) {
                for (int j = 0; j < WARPS; j++) {
                    RTile ra = dstate[j];
                    ((float4*)share)[j*32+tid] = *((float4*)ra.data);
                }
            }
            RTile dstatei;// = dstate[i*WARPS+warpi];
            __syncthreads();
            *((float4*)dstatei.data) = ((float4*)share)[warpi*32+tid];
            __syncthreads();
            RTile dstatei_t = transpose(dstatei);
            v_dstate += from_warp(transpose(vt), i, (float4*)share) % dstatei_t;
            u_dstate += from_warp(transpose(ut), i, (float4*)share) % dstatei_t;
            dw += ones % ((RTile)state[i*WARPS+warpi].t()*dstatei_t);
        }

        FTile db = fw * u_dstate;
        db += transpose(dab) % wat;
        db += transpose(dqb) % wqt;
        db = inv_incl_pref * db;
        GTile(db_+off, stride) = RTile(db);

        FTile dk2 = fw * v_dstate;
        dk2 += transpose(dak) % wat;
        dk2 += transpose(dqk) % wqt;
        dk += inv_incl_pref * dk2;
        GTile(dk_+off, stride) = RTile(dk);

        dw = fw * dw;
        dw += fast_dw<1>(dab,wa,bwi);
        dw += fast_dw<1>(dak,wa,kwi);
        dw += fast_dw<0>(dqb,wq,bwi);
        dw += fast_dw<0>(dqk,wq,kwi);
        FTile tmp;
        dw += cumsumv<0,0>(tmp = v_dstate*(fw*kwi));
        dw += cumsumv<0,0>(tmp = u_dstate*(fw*bwi));
        dw += cumsumv<0,1>(tmp = dab_u_state0*wa);
        dw += cumsumv<1,1>(tmp = dy_state0*wq);

        FTile dw_fac = (RTile)sw;
        apply_(dw_fac, [](float x) { return -__expf(x); });
        dw = dw * dw_fac;
        GTile(dw_+off, stride) = RTile(dw);

        for (int i = 0; i < WARPS; i++) {
            FTile ndstate = dstate[i] * from_warp(fw, i, (float4*)share);
            ndstate += dyt % from_warp(wqt, i, (float4*)share);
            ndstate += dab_ut % from_warp(wat, i, (float4*)share);
            dstate[i] = ndstate;
        }
    }
    for (int i = 0; i < WARPS; i++) {
        int off = bi*H*C*C + hi*C*C + warpi*16*C + i*16;
        GTile(ds0_+off, C) = dstate[i];
    }

    int j0 = threadIdx.x%4*2+threadIdx.x/32*16;
    for (int i = 0; i < 12; i++) {
        dparams[i] += __shfl_xor_sync(0xffffffff, dparams[i], 4);
        dparams[i] += __shfl_xor_sync(0xffffffff, dparams[i], 8);
        dparams[i] += __shfl_xor_sync(0xffffffff, dparams[i], 16);
    }
    if (threadIdx.x%32 < 4) {
        for (int i = 0; i < 4; i++) {
            int j = hi*C+j0+i%2+i/2*8;
            atomicAdd(&dparams_[j], dparams[i]);
            atomicAdd(&dparams_[H*C+j], dparams[i+4]);
            atomicAdd(&dparams_[H*C*2+j], dparams[i+8]);
        }
    }
}

void cuda_backward(int B, int T, int H, bf*w, bf*q, bf*k, bf*v, bf*a, bf*b, bf*g, float*params, bf*dy, bf*s, bf*dsT, bf*dw, bf*dq, bf*dk, bf*dv, bf*da, bf*db, bf*dg, float*dparams, bf*ds0) {
    assert(T%16 == 0);
    constexpr int tmp_size1 = sizeof(float)*32*8*WARPS, tmp_size2 = sizeof(float)*16*16*2;
    constexpr int threads = 32*WARPS, shared_mem = sizeof(STile)*WARPS*bw_stages*(8+WARPS) + (tmp_size1 > tmp_size2 ? tmp_size1 : tmp_size2);
    static int reported = 0;
    if (!reported++) {
#if defined VERBOSE
        printf("backward_kernel() uses %d bytes of (dynamic) shared memory\n", shared_mem);
#endif
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(backward_kernel));
        int cur_mem = attr.maxDynamicSharedSizeBytes;
        if (shared_mem > cur_mem) {
#if defined VERBOSE
            printf("Increasing backward_kernel's MaxDynamicSharedMemorySize from %d to %d\n", cur_mem, shared_mem);
#endif
            assert(!hipFuncSetAttribute(reinterpret_cast<const void*>(backward_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem));
        }
    }
    backward_kernel<<<dim3(H,B), dim3(threads), shared_mem>>>(T,H,w,q,k,v,a,b,g,params,dy,s,dsT,dw,dq,dk,dv,da,db,dg,dparams,ds0);
}
